#include "hip/hip_runtime.h"
#include "fmm.h"

unsigned int hostOffsetSize;
unsigned int hostAccelSize;
unsigned int hostPosTargetSize;
unsigned int hostPosSourceSize;
unsigned int hostMnmTargetSize;
unsigned int hostMnmSourceSize;
unsigned int hostLnmTargetSize;
unsigned int hostLnmSourceSize;
unsigned int hostConstantSize;

int *hostOffset;
float3 *hostAccel;
float3 *hostPosTarget;
float4 *hostPosSource;
float *hostMnmTarget;
float *hostMnmSource;
float *hostLnmTarget;
float *hostLnmSource;
float *hostConstant;

static unsigned int is_set=0;
static unsigned int deviceOffsetSize=0;
static unsigned int deviceAccelSize=0;
static unsigned int devicePosTargetSize=0;
static unsigned int devicePosSourceSize=0;
static unsigned int deviceMnmTargetSize=0;
static unsigned int deviceMnmSourceSize=0;
static unsigned int deviceLnmTargetSize=0;
static unsigned int deviceLnmSourceSize=0;

static int *deviceOffset;
static float3 *deviceAccel;
static float3 *devicePosTarget;
static float4 *devicePosSource;
static float *deviceMnmTarget;
static float *deviceMnmSource;
static float *deviceLnmTarget;
static float *deviceLnmSource;

__device__ __constant__ float deviceConstant[4];

#include "gpukernelcore_p4.cu"

double get_gpu_time(void)
{
  struct timeval tv;
  struct timezone tz;
  if (is_set==1) hipDeviceSynchronize();
  gettimeofday(&tv, &tz);
  return ((double)(tv.tv_sec+tv.tv_usec*1.0e-6));
}

FmmSystem tree;

// direct summation kernel
void FmmKernel::direct(int n) {
  int i,nicall,njcall,icall,iwork1,iwork2,ista,iend,ibase,isize,iblok,is,im;
  int jcall,jwork1,jwork2,jsta,jend,jbase,jsize;
  int nj,nflop;
  const int offsetStride = 2*maxP2PInteraction+1;
  double tic,toc,flops,t[10],op=0;

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  for( i=0; i<n; i++ ) {
    bodyAccel[i].x = 0;
    bodyAccel[i].y = 0;
    bodyAccel[i].z = 0;
  }
  nicall = n/targetBufferSize+1;
  njcall = n/sourceBufferSize+1;
  iblok = (n/nicall+threadsPerBlockTypeA-1)/threadsPerBlockTypeA;

  hostOffsetSize=sizeof(int)*iblok*offsetStride;
  hostPosTargetSize=sizeof(float3)*targetBufferSize;
  hostPosSourceSize=sizeof(float4)*sourceBufferSize;
  hostAccelSize=sizeof(float3)*targetBufferSize;

  hostOffset=(int *)malloc(hostOffsetSize);
  hostPosTarget=(float3 *)malloc(hostPosTargetSize);
  hostPosSource=(float4 *)malloc(hostPosSourceSize);
  hostAccel=(float3 *)malloc(hostAccelSize);

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostPosTargetSize>devicePosTargetSize) {
    if(devicePosTargetSize!=0) CUDA_SAFE_CALL(hipFree(devicePosTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosTarget,hostPosTargetSize));
    devicePosTargetSize=hostPosTargetSize;
  }
  if (hostPosSourceSize>devicePosSourceSize) {
    if(devicePosSourceSize!=0) CUDA_SAFE_CALL(hipFree(devicePosSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosSource,hostPosSourceSize));
    devicePosSourceSize=hostPosSourceSize;
  }
  if (hostAccelSize>deviceAccelSize) {
    if(deviceAccelSize!=0) CUDA_SAFE_CALL(hipFree(deviceAccel));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceAccel,hostAccelSize));
    deviceAccelSize=hostAccelSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  if( n < 1 ) nicall = 0;
  for( icall=0; icall<nicall; icall++ ) {
    iwork1 = n/nicall;
    iwork2 = n%nicall;
    ista = icall*iwork1+std::min(icall,iwork2);
    iend = ista+iwork1-1;
    if( iwork2 > icall ) iend++;
    ibase = ista;
    isize = iend-ibase+1;
    iblok = 0;
    for( is=0; is<isize; is+=threadsPerBlockTypeA ) {
      for( i=0; i<std::min(isize-is,threadsPerBlockTypeA); i++ ) {
        im = iblok*threadsPerBlockTypeA+i;
        hostPosTarget[im] = *(float3*) &bodyPos[ibase+is+i];
      }
      for( i=isize-is; i<threadsPerBlockTypeA; i++ ) {
        im = iblok*threadsPerBlockTypeA+i;
        hostPosTarget[im].x = 0;
        hostPosTarget[im].y = 0;
        hostPosTarget[im].z = 0;
      }
      iblok++;
    }
    for( jcall=0; jcall<njcall; jcall++ ) {
      jwork1 = n/njcall;
      jwork2 = n%njcall;
      jsta = jcall*jwork1+std::min(jcall,jwork2);
      jend = jsta+jwork1;
      if( jwork2 > jcall ) jend++;
      jbase = jsta;
      jsize = jend-jbase;
      for( i=0; i<iblok; i++ ) {
        hostOffset[i*offsetStride] = 1;
        hostOffset[i*offsetStride+1] = 0;
        hostOffset[i*offsetStride+2] = jsize;
      }
      for( i=jsta; i<jend; i++ ) {
        nj = i-jsta;
        hostPosSource[nj] = *(float4*) &bodyPos[i];
      }
      nj++;
      op += (double) isize*jsize;

      toc=tic;
      tic=get_gpu_time();
      t[0]+=tic-toc;
      CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(devicePosTarget,hostPosTarget,hostPosTargetSize,hipMemcpyHostToDevice));
      CUDA_SAFE_CALL(hipMemcpy(devicePosSource,hostPosSource,hostPosSourceSize,hipMemcpyHostToDevice));
      toc=tic;
      tic=get_gpu_time();
      t[2]+=tic-toc;

      dim3 block(threadsPerBlockTypeA);
      dim3 grid(iblok);
      p2p_kernel<<< grid, block >>>(deviceOffset,devicePosTarget,devicePosSource,deviceAccel);
      cudaCheckError();
      nflop = 19;

      toc=tic;
      tic=get_gpu_time();
      t[3]+=tic-toc;
      CUDA_SAFE_CALL(hipMemcpy(hostAccel,deviceAccel,hostAccelSize,hipMemcpyDeviceToHost));
      toc=tic;
      tic=get_gpu_time();
      t[2]+=tic-toc;

      iblok = 0;
      for( is=0; is<isize; is+=threadsPerBlockTypeA ) {
        for( i=0; i<std::min(isize-is,threadsPerBlockTypeA); i++ ) {
          im = iblok*threadsPerBlockTypeA+i;
          bodyAccel[ibase+is+i].x += inv4PI*hostAccel[im].x;
          bodyAccel[ibase+is+i].y += inv4PI*hostAccel[im].y;
          bodyAccel[ibase+is+i].z += inv4PI*hostAccel[im].z;
        }
        iblok++;
      }
    }
  }
  free(hostOffset);
  free(hostPosTarget);
  free(hostPosSource);
  free(hostAccel);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("p2p hipMalloc : %f s\n",t[1]);
//  printf("p2p hipMemcpy : %f s\n",t[2]);
//  printf("p2p cudaKernel : %f s\n",t[3]);
//  printf("p2p other      : %f s\n",t[0]);
//  printf("p2p flops      : %f G\n",flops/1e9);
  tic=flops;
}

// precalculate M2L translation matrix and Wigner rotation matrix
void FmmKernel::precalc() {
  int i,j;

  for( j=0; j<numBoxIndexTotal; j++ ) {
    for( i=0; i<numCoefficients; i++ ) {
      Mnm[j][i] = 0;
    }
  }
}

// p2p
void FmmKernel::p2p(int numBoxIndex) {
  int nicall,jc,jj,ii,njd,ij,icall,jcall,iblok,im,jjd,j,ibase,isize,is,i,ijc,jjdd;
  int ni,nj,nflop,*jbase,*jsize,*njcall,*njj,**interactionListOffsetStart,**interactionListOffsetEnd;
  const int offsetStride = 2*maxP2PInteraction+1;
  double tic,toc,flops,t[10],op=0;

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeA*offsetStride;
  hostPosTargetSize=sizeof(float3)*targetBufferSize;
  hostPosSourceSize=sizeof(float4)*sourceBufferSize;
  hostAccelSize=sizeof(float3)*targetBufferSize;

  hostOffset=(int *)malloc(hostOffsetSize);
  hostPosTarget=(float3 *)malloc(hostPosTargetSize);
  hostPosSource=(float4 *)malloc(hostPosSourceSize);
  hostAccel=(float3 *)malloc(hostAccelSize);
  interactionListOffsetStart = new int* [maxM2LInteraction];
  for( i=0; i<maxM2LInteraction; i++ ) interactionListOffsetStart[i] = new int [numBoxIndexLeaf];
  interactionListOffsetEnd = new int* [maxM2LInteraction];
  for( i=0; i<maxM2LInteraction; i++ ) interactionListOffsetEnd[i] = new int [numBoxIndexLeaf];
  jbase = new int [numBoxIndexLeaf];
  jsize = new int [numBoxIndexLeaf];
  njcall = new int [numBoxIndexLeaf];
  njj = new int [numBoxIndexLeaf];

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostPosTargetSize>devicePosTargetSize) {
    if(devicePosTargetSize!=0) CUDA_SAFE_CALL(hipFree(devicePosTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosTarget,hostPosTargetSize));
    devicePosTargetSize=hostPosTargetSize;
  }
  if (hostPosSourceSize>devicePosSourceSize) {
    if(devicePosSourceSize!=0) CUDA_SAFE_CALL(hipFree(devicePosSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosSource,hostPosSourceSize));
    devicePosSourceSize=hostPosSourceSize;
  }
  if (hostAccelSize>deviceAccelSize) {
    if(deviceAccelSize!=0) CUDA_SAFE_CALL(hipFree(deviceAccel));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceAccel,hostAccelSize));
    deviceAccelSize=hostAccelSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  ni = 0;
  nj = 0;
  nicall = 0;
  boxOffsetStart[0] = 0;
  jc = 0;
  for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    if( numInteraction[ii] != 0 ) {
      njd = 0;
      jc = 0;
      interactionListOffsetStart[0][ii] = 0;
      for( ij=0; ij<numInteraction[ii]; ij++ ) {
        jj = interactionList[ii][ij];
        if( njj[jj] == 0 ) {
          nj += particleOffset[1][jj]-particleOffset[0][jj]+1;
          njj[jj] = 1;
        }
        njd += particleOffset[1][jj]-particleOffset[0][jj]+1;
        if( njd > sourceBufferSize ) {
          interactionListOffsetEnd[jc][ii] = ij-1;
          jc++;
          interactionListOffsetStart[jc][ii] = ij;
          njd = particleOffset[1][jj]-particleOffset[0][jj]+1;
        }
      }
      interactionListOffsetEnd[jc][ii] = numInteraction[ii]-1;
      ni += ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeA)/threadsPerBlockTypeA+1)
            *threadsPerBlockTypeA;
      if( jc != 0 ) {
        if( ii > boxOffsetStart[nicall] ) {
          njcall[nicall] = 1;
          boxOffsetEnd[nicall] = ii-1;
          nicall++;
          assert( nicall < numBoxIndexLeaf );
          boxOffsetStart[nicall] = ii;
          for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
        }
        if( ii != numBoxIndex ) {
          njcall[nicall] = jc+1;
          boxOffsetEnd[nicall] = ii;
          nicall++;
          assert( nicall < numBoxIndexLeaf );
          boxOffsetStart[nicall] = ii+1;
          for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
          ni = 0;
          nj = 0;
        }
      } else if ( ni > targetBufferSize || nj > sourceBufferSize ) {
        njcall[nicall] = jc+1;
        boxOffsetEnd[nicall] = ii-1;
        nicall++;
        assert( nicall < numBoxIndexLeaf );
        boxOffsetStart[nicall] = ii;
        for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
        ni = ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeA)/threadsPerBlockTypeA+1)
             *threadsPerBlockTypeA;
        nj = 0;
        for( ij=0; ij<numInteraction[ii]; ij++ ) {
          jj = interactionList[ii][ij];
          nj += particleOffset[1][jj]-particleOffset[0][jj]+1;
          njj[jj] = 1;
        }
      }
    }
  }
  njcall[nicall] = jc+1;
  boxOffsetEnd[nicall] = numBoxIndex-1;
  if(numBoxIndex != 0) nicall++;

  for( icall=0; icall<nicall; icall++ ) {
    for( jcall=0; jcall<njcall[icall]; jcall++ ) {
      iblok = 0;
      jc = 0;
      jjd = 0;
      for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
      for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
        if( numInteraction[ii] != 0 ) {
          for( ij=interactionListOffsetStart[jcall][ii]; ij<=interactionListOffsetEnd[jcall][ii]; ij++ ) {
            jj = interactionList[ii][ij];
            if( njj[jj] == 0 ) {
              jbase[jjd] = jc;
              for( j=particleOffset[0][jj]; j<=particleOffset[1][jj]; j++ ) {
                hostPosSource[jc] = *(float4*) &bodyPos[j];
                jc++;
              }
              jsize[jjd] = jc-jbase[jjd];
              jjd++;
              njj[jj] = jjd;
            }
          }
          ibase = particleOffset[0][ii];
          isize = particleOffset[1][ii]-ibase+1;
          for( is=0; is<isize; is+=threadsPerBlockTypeA ) {
            for( i=0; i<std::min(isize-is,threadsPerBlockTypeA); i++ ) {
              im = iblok*threadsPerBlockTypeA+i;
              hostPosTarget[im] = *(float3*) &bodyPos[ibase+is+i];
            }
            for( i=isize-is; i<threadsPerBlockTypeA; i++ ) {
              im = iblok*threadsPerBlockTypeA+i;
              hostPosTarget[im].x = 0;
              hostPosTarget[im].y = 0;
              hostPosTarget[im].z = 0;
            }
            hostOffset[iblok*offsetStride] = interactionListOffsetEnd[jcall][ii]
                                            -interactionListOffsetStart[jcall][ii]+1;
            ijc = 0;
            for( ij=interactionListOffsetStart[jcall][ii]; ij<=interactionListOffsetEnd[jcall][ii]; ij++ ) {
              jj = interactionList[ii][ij];
              if( njj[jj] != 0 ) {
                jjdd = njj[jj]-1;
                hostOffset[iblok*offsetStride+2*ijc+1] = jbase[jjdd];
                hostOffset[iblok*offsetStride+2*ijc+2] = jsize[jjdd];
                op += (double) threadsPerBlockTypeA*jsize[jjdd];
                ijc++;
              }
            }
            iblok++;
          }
        }
      }

      if( iblok != 0 ) {
        toc=tic;
        tic=get_gpu_time();
        t[0]+=tic-toc;
        CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(devicePosTarget,hostPosTarget,hostPosTargetSize,hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(devicePosSource,hostPosSource,hostPosSourceSize,hipMemcpyHostToDevice));
        toc=tic;
        tic=get_gpu_time();
        t[2]+=tic-toc;

        dim3 block(threadsPerBlockTypeA);
        dim3 grid(iblok);
        p2p_kernel<<< grid, block >>>(deviceOffset,devicePosTarget,devicePosSource,deviceAccel);
        cudaCheckError();
        nflop = 19;

        toc=tic;
        tic=get_gpu_time();
        t[3]+=tic-toc;
        CUDA_SAFE_CALL(hipMemcpy(hostAccel,deviceAccel,hostAccelSize,hipMemcpyDeviceToHost));
        toc=tic;
        tic=get_gpu_time();
        t[2]+=tic-toc;

      }
      iblok = 0;
      for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
        if( numInteraction[ii] != 0 ) {
          ibase = particleOffset[0][ii];
          isize = particleOffset[1][ii]-ibase+1;
          for( is=0; is<isize; is+=threadsPerBlockTypeA ) {
            for( i=0; i<std::min(isize-is,threadsPerBlockTypeA); i++ ) {
              im = iblok*threadsPerBlockTypeA+i;
              bodyAccel[ibase+is+i].x += inv4PI*hostAccel[im].x;
              bodyAccel[ibase+is+i].y += inv4PI*hostAccel[im].y;
              bodyAccel[ibase+is+i].z += inv4PI*hostAccel[im].z;
            }
            iblok++;
          }
        }
      }
    }
  }
  free(hostOffset);
  free(hostPosTarget);
  free(hostPosSource);
  free(hostAccel);
  for( i=0; i<maxM2LInteraction; i++ ) delete[] interactionListOffsetStart[i];
  delete[] interactionListOffsetStart;
  for( i=0; i<maxM2LInteraction; i++ ) delete[] interactionListOffsetEnd[i];
  delete[] interactionListOffsetEnd;
  delete[] jbase;
  delete[] jsize;
  delete[] njcall;
  delete[] njj;

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("p2p hipMalloc : %f s\n",t[1]);
//  printf("p2p hipMemcpy : %f s\n",t[2]);
//  printf("p2p cudaKernel : %f s\n",t[3]);
//  printf("p2p other      : %f s\n",t[0]);
//  printf("p2p flops      : %f G\n",flops/1e9);
  tic=flops;
}

// p2m
void FmmKernel::p2m(int numBoxIndex) {
  int ncall,jj,icall,iblok,jc,jbase,j,jsize,jm;
  int i,ni,nj,nflop;
  vec3<int> boxIndex3D;
  const int offsetStride = 5;
  double tic,toc,flops,t[10],boxSize,op=0;

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << maxLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostMnmTargetSize=sizeof(float)*2*targetBufferSize;
  hostPosSourceSize=sizeof(float4)*sourceBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostMnmTarget=(float *)malloc(hostMnmTargetSize);
  hostPosSource=(float4 *)malloc(hostPosSourceSize);

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=(float) boxMin.x;
  hostConstant[2]=(float) boxMin.y;
  hostConstant[3]=(float) boxMin.z;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostMnmTargetSize>deviceMnmTargetSize) {
    if(deviceMnmTargetSize!=0) CUDA_SAFE_CALL(hipFree(deviceMnmTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceMnmTarget,hostMnmTargetSize));
    deviceMnmTargetSize=hostMnmTargetSize;
  }
  if (hostPosSourceSize>devicePosSourceSize) {
    if(devicePosSourceSize!=0) CUDA_SAFE_CALL(hipFree(devicePosSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosSource,hostPosSourceSize));
    devicePosSourceSize=hostPosSourceSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  ni = 0;
  nj = 0;
  ncall = 0;
  boxOffsetStart[0] = 0;
  for( jj=0; jj<numBoxIndex; jj++ ) {
    ni += ((numCoefficients+threadsPerBlockTypeB)/threadsPerBlockTypeB+1)*threadsPerBlockTypeB;
    nj += particleOffset[1][jj]-particleOffset[0][jj]+1;
    if( ni > targetBufferSize || nj > sourceBufferSize ) {
      boxOffsetEnd[ncall] = jj-1;
      ncall++;
      boxOffsetStart[ncall] = jj;
      ni = ((numCoefficients+threadsPerBlockTypeB)/threadsPerBlockTypeB+1)*threadsPerBlockTypeB;
      nj = particleOffset[1][jj]-particleOffset[0][jj]+1;
    }
  }
  boxOffsetEnd[ncall] = numBoxIndex-1;
  if(numBoxIndex != 0) ncall++;

  for( icall=0; icall<ncall; icall++ ) {
    iblok = 0;
    jc = 0;
    for( jj=boxOffsetStart[icall]; jj<=boxOffsetEnd[icall]; jj++ ) {
      jbase = jc;
      for( j=particleOffset[0][jj]; j<=particleOffset[1][jj]; j++ ) {
        hostPosSource[jc] = *(float4*) &bodyPos[j];
        jc++;
      }
      jsize = jc-jbase;
      tree.unmorton(boxIndexFull[jj],boxIndex3D);
      hostOffset[iblok*offsetStride] = jbase;
      hostOffset[iblok*offsetStride+1] = jsize;
      hostOffset[iblok*offsetStride+2] = boxIndex3D.x;
      hostOffset[iblok*offsetStride+3] = boxIndex3D.y;
      hostOffset[iblok*offsetStride+4] = boxIndex3D.z;
      op += threadsPerBlockTypeB*jsize;
      iblok++;
    }

    toc=tic;
    tic=get_gpu_time();
    t[0]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
    CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devicePosSource,hostPosSource,hostPosSourceSize,hipMemcpyHostToDevice));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    dim3 block(threadsPerBlockTypeB);
    dim3 grid(iblok);
    p2m_kernel<<< grid, block >>>(deviceOffset,deviceMnmTarget,devicePosSource);
    cudaCheckError();
    nflop = 20*numCoefficients;

    toc=tic;
    tic=get_gpu_time();
    t[3]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpy(hostMnmTarget,deviceMnmTarget,hostMnmTargetSize,hipMemcpyDeviceToHost));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    iblok = 0;
    for( jj=boxOffsetStart[icall]; jj<=boxOffsetEnd[icall]; jj++ ) {
      for( j=0; j<numCoefficients; j++ ) {
        jm = iblok*threadsPerBlockTypeB+j;
        Mnm[jj][j] = std::complex<double>(hostMnmTarget[2*jm+0],hostMnmTarget[2*jm+1]);
      }
      iblok++;
    }
  }

  free(hostConstant);
  free(hostOffset);
  free(hostMnmTarget);
  free(hostPosSource);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("p2m hipMalloc : %f s\n",t[1]);
//  printf("p2m hipMemcpy : %f s\n",t[2]);
//  printf("p2m cudaKernel : %f s\n",t[3]);
//  printf("p2m other      : %f s\n",t[0]);
//  printf("p2m flops      : %f G\n",flops/1e9);
  tic=flops;
}

// m2m
void FmmKernel::m2m(int numBoxIndex, int numBoxIndexOld, int numLevel) {
  int ii,ib,j,ncall,jj,icall;
  int iblok,jc,nfic,jb,jbase,jsize,nfip,jm;
  int i,nj,nflop;
  vec3<int> boxIndex3D;
  const int offsetStride = 5;
  double tic,toc,flops,t[10],boxSize,op=0;
  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << numLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostMnmTargetSize=sizeof(float)*2*targetBufferSize;
  hostMnmSourceSize=sizeof(float)*2*sourceBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostMnmTarget=(float *)malloc(hostMnmTargetSize);
  hostMnmSource=(float *)malloc(hostMnmSourceSize);

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=0;
  hostConstant[2]=0;
  hostConstant[3]=0;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostMnmTargetSize>deviceMnmTargetSize) {
    if(deviceMnmTargetSize!=0) CUDA_SAFE_CALL(hipFree(deviceMnmTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceMnmTarget,hostMnmTargetSize));
    deviceMnmTargetSize=hostMnmTargetSize;
  }
  if (hostMnmSourceSize>deviceMnmSourceSize) {
    if(deviceMnmSourceSize!=0) CUDA_SAFE_CALL(hipFree(deviceMnmSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceMnmSource,hostMnmSourceSize));
    deviceMnmSourceSize=hostMnmSourceSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  for( ii=0; ii<numBoxIndex; ii++ ) {
    ib = ii+levelOffset[numLevel-1];
    for( j=0; j<numCoefficients; j++ ) {
      Mnm[ib][j] = 0;
    }
  }

  nj = 0;
  ncall = 0;
  boxOffsetStart[0] = 0;
  for( jj=0; jj<numBoxIndexOld; jj++ ) {
    nj += threadsPerBlockTypeB;
    if( nj > sourceBufferSize ) {
      boxOffsetEnd[ncall] = jj-1;
      ncall++;
      boxOffsetStart[ncall] = jj;
      nj = threadsPerBlockTypeB;
    }
  }
  boxOffsetEnd[ncall] = numBoxIndexOld-1;
  if(numBoxIndexOld != 0) ncall++;

  for( icall=0; icall<ncall; icall++ ) {
    iblok = 0;
    jc = 0;
    for( jj=boxOffsetStart[icall]; jj<=boxOffsetEnd[icall]; jj++ ) {
      jb = jj+levelOffset[numLevel];
      nfic = boxIndexFull[jb]%8;
      tree.unmorton(nfic,boxIndex3D);
      jbase = jc;
      for( j=0; j<numCoefficients; j++ ) {
        hostMnmSource[2*jc+0] = std::real(Mnm[jb][j]);
        hostMnmSource[2*jc+1] = std::imag(Mnm[jb][j]);
        jc++;
      }
      jsize = jc-jbase;
      hostOffset[iblok*offsetStride+0] = 1;
      hostOffset[iblok*offsetStride+1] = jbase;
      hostOffset[iblok*offsetStride+2] = 1-boxIndex3D.x*2;
      hostOffset[iblok*offsetStride+3] = 1-boxIndex3D.y*2;
      hostOffset[iblok*offsetStride+4] = 1-boxIndex3D.z*2;
      op += threadsPerBlockTypeB*jsize;
      iblok++;
    }

    toc=tic;
    tic=get_gpu_time();
    t[0]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
    CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(deviceMnmSource,hostMnmSource,hostMnmSourceSize,hipMemcpyHostToDevice));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    dim3 block(threadsPerBlockTypeB);
    dim3 grid(iblok);
    m2m_kernel<<< grid, block >>>(deviceOffset,deviceMnmTarget,deviceMnmSource);
    cudaCheckError();
    nflop = 48;

    toc=tic;
    tic=get_gpu_time();
    t[3]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpy(hostMnmTarget,deviceMnmTarget,hostMnmTargetSize,hipMemcpyDeviceToHost));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    iblok = 0;
    for( jj=boxOffsetStart[icall]; jj<=boxOffsetEnd[icall]; jj++ ) {
      jb = jj+levelOffset[numLevel];
      nfip = boxIndexFull[jb]/8;
      ib = boxIndexMask[nfip]+levelOffset[numLevel-1];
      for( j=0; j<numCoefficients; j++ ) {
        jm = iblok*threadsPerBlockTypeB+j;
        Mnm[ib][j] += std::complex<double>(hostMnmTarget[2*jm+0],hostMnmTarget[2*jm+1]);
      }
      iblok++;
    }
  }

  free(hostConstant);
  free(hostOffset);
  free(hostMnmTarget);
  free(hostMnmSource);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("m2m hipMalloc : %f s\n",t[1]);
//  printf("m2m hipMemcpy : %f s\n",t[2]);
//  printf("m2m cudaKernel : %f s\n",t[3]);
//  printf("m2m other      : %f s\n",t[0]);
//  printf("m2m flops      : %f G\n",flops/1e9);
  tic=flops;

}

// m2l
void FmmKernel::m2l(int numBoxIndex, int numLevel) {
  int i,j,ncall,jj,ii,ib,ij,icall,iblok,jc,jjd;
  int jb,jbd,ix,iy,iz,is,jjdd,jx,jy,jz,isize,im;
  int ni,nj,nflop,*jbase,*jsize,*njj;
  vec3<int> boxIndex3D;
  const int offsetStride = 4*maxM2LInteraction+1;
  double tic,toc,flops,t[10],boxSize,op;
  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << numLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostLnmTargetSize=sizeof(float)*2*targetBufferSize;
  hostMnmSourceSize=sizeof(float)*2*sourceBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostLnmTarget=(float *)malloc(hostLnmTargetSize);
  hostMnmSource=(float *)malloc(hostMnmSourceSize);
  jbase = new int [numBoxIndexLeaf];
  jsize = new int [numBoxIndexLeaf];
  njj = new int [numBoxIndexLeaf];

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=0;
  hostConstant[2]=0;
  hostConstant[3]=0;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostLnmTargetSize>deviceLnmTargetSize) {
    if(deviceLnmTargetSize!=0) CUDA_SAFE_CALL(hipFree(deviceLnmTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceLnmTarget,hostLnmTargetSize));
    deviceLnmTargetSize=hostLnmTargetSize;
  }
  if (hostMnmSourceSize>deviceMnmSourceSize) {
    if(deviceMnmSourceSize!=0) CUDA_SAFE_CALL(hipFree(deviceMnmSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceMnmSource,hostMnmSourceSize));
    deviceMnmSourceSize=hostMnmSourceSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  if( numLevel == 2 ) {
    for( i=0; i<numBoxIndex; i++ ) {
      for( j=0; j<numCoefficients; j++ ) {
        Lnm[i][j] = 0;
      }
    }
  }

  ni = 0;
  nj = 0;
  ncall = 0;
  boxOffsetStart[0] = 0;
  for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    ni += ((numCoefficients+threadsPerBlockTypeB)/threadsPerBlockTypeB)*threadsPerBlockTypeB;
    if( numInteraction[ii] != 0 ) {
      for( ij=0; ij<numInteraction[ii]; ij++ ) {
        jj = interactionList[ii][ij];
        if( njj[jj] == 0 ) {
          nj += numCoefficients;
          njj[jj] = 1;
        }
      }
    }
    if ( ni > targetBufferSize || nj > sourceBufferSize ) {
      boxOffsetEnd[ncall] = ii-1;
      ncall++;
      boxOffsetStart[ncall] = ii;
      ni = ((numCoefficients+threadsPerBlockTypeB)/threadsPerBlockTypeB)*threadsPerBlockTypeB;
      nj = 0;
      for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
      for( ij=0; ij<numInteraction[ii]; ij++ ) {
        jj = interactionList[ii][ij];
        nj += numCoefficients;
        njj[jj] = 1;
      }
    }
  }
  boxOffsetEnd[ncall] = numBoxIndex-1;
  if(numBoxIndex != 0) ncall++;

  for( icall=0; icall<ncall; icall++ ) {
    iblok = 0;
    jc = 0;
    jjd = 0;
    op = 0;
    for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      for( ij=0; ij<numInteraction[ii]; ij++ ) {
        jj = interactionList[ii][ij];
        jb = jj+levelOffset[numLevel-1];
        if( njj[jj] == 0 ) {
          jbase[jjd] = jc;
          for( j=0; j<numCoefficients; j++ ) {
            hostMnmSource[2*jc+0] = std::real(Mnm[jb][j]);
            hostMnmSource[2*jc+1] = std::imag(Mnm[jb][j]);
            jc++;
          }
          jsize[jjd] = jc-jbase[jjd];
          jjd++;
          njj[jj] = jjd;
        }
      }
      ib = ii+levelOffset[numLevel-1];
      tree.unmorton(boxIndexFull[ib],boxIndex3D);
      ix = boxIndex3D.x;
      iy = boxIndex3D.y;
      iz = boxIndex3D.z;
      isize = numCoefficients;
      for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
        hostOffset[iblok*offsetStride] = numInteraction[ii];
        for( ij=0; ij<numInteraction[ii]; ij++ ) {
          jj = interactionList[ii][ij];
          jbd = jj+levelOffset[numLevel-1];
          jjdd = njj[jj]-1;
          tree.unmorton(boxIndexFull[jbd],boxIndex3D);
          jx = boxIndex3D.x;
          jy = boxIndex3D.y;
          jz = boxIndex3D.z;
          hostOffset[iblok*offsetStride+4*ij+1] = jbase[jjdd];
          hostOffset[iblok*offsetStride+4*ij+2] = ix-jx;
          hostOffset[iblok*offsetStride+4*ij+3] = iy-jy;
          hostOffset[iblok*offsetStride+4*ij+4] = iz-jz;
          op += (double) threadsPerBlockTypeB*jsize[jjdd];
        }
        iblok++;
      }
    }

    toc=tic;
    tic=get_gpu_time();
    t[0]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
    CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(deviceMnmSource,hostMnmSource,hostMnmSourceSize,hipMemcpyHostToDevice));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    dim3 block(threadsPerBlockTypeB);
    dim3 grid(iblok);
    m2l_kernel<<< grid, block >>>(deviceOffset,deviceLnmTarget,deviceMnmSource);
    cudaCheckError();
    nflop = 48;

    toc=tic;
    tic=get_gpu_time();
    t[3]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpy(hostLnmTarget,deviceLnmTarget,hostLnmTargetSize,hipMemcpyDeviceToHost));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    iblok = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      isize = numCoefficients;
      for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
        for( i=0; i<std::min(isize-is,threadsPerBlockTypeB); i++ ) {
          im = iblok*threadsPerBlockTypeB+i;
          Lnm[ii][is+i] += std::complex<double>(hostLnmTarget[2*im+0],hostLnmTarget[2*im+1]);
        }
        iblok++;
      }
    }
  }
  for( jj=0; jj<numBoxIndex; jj++ ) {
    jb = jj+levelOffset[numLevel-1];
    for( j=0; j<numCoefficients; j++ ) {
      Mnm[jb][j] = 0;
    }
  }

  free(hostConstant);
  free(hostOffset);
  free(hostLnmTarget);
  free(hostMnmSource);
  delete[] jbase;
  delete[] jsize;
  delete[] njj;

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("m2l hipMalloc : %f s\n",t[1]);
//  printf("m2l hipMemcpy : %f s\n",t[2]);
//  printf("m2l cudaKernel : %f s\n",t[3]);
//  printf("m2l other      : %f s\n",t[0]);
//  printf("m2l flops      : %f G\n",flops/1e9);
  tic=flops;

}

// l2l
void FmmKernel::l2l(int numBoxIndex, int numLevel) {
  int numBoxIndexOld,ii,i;
  int ncall,icall,iblok,ic,nfip,nfic,ib,jbase,jsize,im;
  int ni,nflop;
  vec3<int> boxIndex3D;
  const int offsetStride = 5;
  double tic,toc,flops,t[10],boxSize,op=0;
  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << numLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostLnmTargetSize=sizeof(float)*2*targetBufferSize;
  hostLnmSourceSize=sizeof(float)*2*sourceBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostLnmTarget=(float *)malloc(hostLnmTargetSize);
  hostLnmSource=(float *)malloc(hostLnmSourceSize);

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=0;
  hostConstant[2]=0;
  hostConstant[3]=0;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostLnmTargetSize>deviceLnmTargetSize) {
    if(deviceLnmTargetSize!=0) CUDA_SAFE_CALL(hipFree(deviceLnmTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceLnmTarget,hostLnmTargetSize));
    deviceLnmTargetSize=hostLnmTargetSize;
  }
  if (hostLnmSourceSize>deviceLnmSourceSize) {
    if(deviceLnmSourceSize!=0) CUDA_SAFE_CALL(hipFree(deviceLnmSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceLnmSource,hostLnmSourceSize));
    deviceLnmSourceSize=hostLnmSourceSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  int nbc,neo[numBoxIndexFull];
  nbc = -1;
  numBoxIndexOld = 0;
  for( i=0; i<numBoxIndexFull; i++ ) neo[i] = -1;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    ib = ii+levelOffset[numLevel-1];
    if( nbc != boxIndexFull[ib]/8 ) {
      nbc = boxIndexFull[ib]/8;
      neo[nbc] = numBoxIndexOld;
      numBoxIndexOld++;
    }
  }

  numBoxIndexOld = numBoxIndex;
  if( numBoxIndexOld < 8 ) numBoxIndexOld = 8;
  for( ii=0; ii<numBoxIndexOld; ii++ ) {
    for( i=0; i<numCoefficients; i++ ) {
      LnmOld[ii][i] = Lnm[ii][i];
    }
  }

  ni = 0;
  ncall = 0;
  boxOffsetStart[0] = 0;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    ni += threadsPerBlockTypeB;
    if( ni > sourceBufferSize ) {
      boxOffsetEnd[ncall] = ii-1;
      ncall++;
      boxOffsetStart[ncall] = ii;
      ni = threadsPerBlockTypeB;
    }
  }
  boxOffsetEnd[ncall] = numBoxIndex-1;
  if(numBoxIndex != 0) ncall++;

  for( icall=0; icall<ncall; icall++ ) {
    iblok = 0;
    ic = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      ib = ii+levelOffset[numLevel-1];
      nfip = boxIndexFull[ib]/8;
      nfic = boxIndexFull[ib]%8;
      tree.unmorton(nfic,boxIndex3D);
      ib = neo[nfip];
      jbase = ic;
      for( i=0; i<numCoefficients; i++ ) {
        hostLnmSource[2*ic+0] = std::real(LnmOld[ib][i]);
        hostLnmSource[2*ic+1] = std::imag(LnmOld[ib][i]);
        ic++;
      }
      jsize = ic-jbase;
      hostOffset[iblok*offsetStride+0] = 1;
      hostOffset[iblok*offsetStride+1] = jbase;
      hostOffset[iblok*offsetStride+2] = boxIndex3D.x*2-1;
      hostOffset[iblok*offsetStride+3] = boxIndex3D.y*2-1;
      hostOffset[iblok*offsetStride+4] = boxIndex3D.z*2-1;
      op += threadsPerBlockTypeB*jsize;
      iblok++;
    }

    toc=tic;
    tic=get_gpu_time();
    t[0]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
    CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(deviceLnmSource,hostLnmSource,hostLnmSourceSize,hipMemcpyHostToDevice));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    dim3 block(threadsPerBlockTypeB);
    dim3 grid(iblok);
    l2l_kernel<<< grid, block >>>(deviceOffset,deviceLnmTarget,deviceLnmSource);
    cudaCheckError();
    nflop = 48;

    toc=tic;
    tic=get_gpu_time();
    t[3]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpy(hostLnmTarget,deviceLnmTarget,hostLnmTargetSize,hipMemcpyDeviceToHost));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    iblok = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      for( i=0; i<numCoefficients; i++ ) {
        im = iblok*threadsPerBlockTypeB+i;
        Lnm[ii][i] = std::complex<double>(hostLnmTarget[2*im+0],hostLnmTarget[2*im+1]);
      }
      iblok++;
    }
  }

  free(hostConstant);
  free(hostOffset);
  free(hostLnmTarget);
  free(hostLnmSource);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("l2l hipMalloc : %f s\n",t[1]);
//  printf("l2l hipMemcpy : %f s\n",t[2]);
//  printf("l2l cudaKernel : %f s\n",t[3]);
//  printf("l2l other      : %f s\n",t[0]);
//  printf("l2l flops      : %f G\n",flops/1e9);
  tic=flops;

}

// l2p
void FmmKernel::l2p(int numBoxIndex) {
  int ncall,ii,icall,iblok,jc,jbase,j,jsize,ibase,isize,is,i,im;
  int ni,nj,nflop;
  vec3<int> boxIndex3D;
  const int offsetStride = 5;
  double tic,toc,flops,t[10],boxSize,op=0;
  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << maxLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostPosTargetSize=sizeof(float3)*targetBufferSize;
  hostLnmSourceSize=sizeof(float)*2*sourceBufferSize;
  hostAccelSize=sizeof(float3)*targetBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostPosTarget=(float3 *)malloc(hostPosTargetSize);
  hostLnmSource=(float *)malloc(hostLnmSourceSize);
  hostAccel=(float3 *)malloc(hostAccelSize);

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=(float) boxMin.x;
  hostConstant[2]=(float) boxMin.y;
  hostConstant[3]=(float) boxMin.z;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostPosTargetSize>devicePosTargetSize) {
    if(devicePosTargetSize!=0) CUDA_SAFE_CALL(hipFree(devicePosTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosTarget,hostPosTargetSize));
    devicePosTargetSize=hostPosTargetSize;
  }
  if (hostLnmSourceSize>deviceLnmSourceSize) {
    if(deviceLnmSourceSize!=0) CUDA_SAFE_CALL(hipFree(deviceLnmSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceLnmSource,hostLnmSourceSize));
    deviceLnmSourceSize=hostLnmSourceSize;
  }
  if (hostAccelSize>deviceAccelSize) {
    if(deviceAccelSize!=0) CUDA_SAFE_CALL(hipFree(deviceAccel));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceAccel,hostAccelSize));
    deviceAccelSize=hostAccelSize;
  }
  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  ni = 0;
  nj = 0;
  ncall = 0;
  boxOffsetStart[0] = 0;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    ni += ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeB)
           /threadsPerBlockTypeB+1)*threadsPerBlockTypeB;
    nj += numCoefficients;
    if( ni > targetBufferSize || nj > sourceBufferSize ) {
      boxOffsetEnd[ncall] = ii-1;
      ncall++;
      boxOffsetStart[ncall] = ii;
      ni = ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeB)
            /threadsPerBlockTypeB+1)*threadsPerBlockTypeB;
      nj = numCoefficients;
    }
  }
  boxOffsetEnd[ncall] = numBoxIndex-1;
  if(numBoxIndex != 0) ncall++;

  for( icall=0; icall<ncall; icall++ ) {
    iblok = 0;
    jc = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      jbase = jc;
      for( j=0; j<numCoefficients; j++ ) {
        hostLnmSource[2*jc+0] = std::real(Lnm[ii][j]);
        hostLnmSource[2*jc+1] = std::imag(Lnm[ii][j]);
        jc++;
      }
      jsize = jc-jbase;
      ibase = particleOffset[0][ii];
      isize = particleOffset[1][ii]-ibase+1;
      for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
        for( i=0; i<std::min(isize-is,threadsPerBlockTypeB); i++ ) {
          im = iblok*threadsPerBlockTypeB+i;
          hostPosTarget[im] = *(float3*) &bodyPos[ibase+is+i];
        }
        for( i=isize-is; i<threadsPerBlockTypeB; i++ ) {
          im = iblok*threadsPerBlockTypeB+i;
          hostPosTarget[im].x = 0;
          hostPosTarget[im].y = 0;
          hostPosTarget[im].z = 0;
        }
        tree.unmorton(boxIndexFull[ii],boxIndex3D);
        hostOffset[iblok*offsetStride+0] = jbase;
        hostOffset[iblok*offsetStride+1] = jsize;
        hostOffset[iblok*offsetStride+2] = boxIndex3D.x;
        hostOffset[iblok*offsetStride+3] = boxIndex3D.y;
        hostOffset[iblok*offsetStride+4] = boxIndex3D.z;
        op += threadsPerBlockTypeB*jsize;
        iblok++;
      }
    }

    toc=tic;
    tic=get_gpu_time();
    t[0]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
    CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devicePosTarget,hostPosTarget,hostPosTargetSize,hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(deviceLnmSource,hostLnmSource,hostLnmSourceSize,hipMemcpyHostToDevice));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    dim3 block(threadsPerBlockTypeB);
    dim3 grid(iblok);
    l2p_kernel<<< grid, block >>>(deviceOffset,devicePosTarget,deviceLnmSource,deviceAccel);
    cudaCheckError();
    nflop = 56;

    toc=tic;
    tic=get_gpu_time();
    t[3]+=tic-toc;
    CUDA_SAFE_CALL(hipMemcpy(hostAccel,deviceAccel,hostAccelSize,hipMemcpyDeviceToHost));
    toc=tic;
    tic=get_gpu_time();
    t[2]+=tic-toc;

    iblok = 0;
    for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
      ibase = particleOffset[0][ii];
      isize = particleOffset[1][ii]-ibase+1;
      for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
        for( i=0; i<std::min(isize-is,threadsPerBlockTypeB); i++ ) {
          im = iblok*threadsPerBlockTypeB+i;
          bodyAccel[ibase+is+i].x += inv4PI*hostAccel[im].x;
          bodyAccel[ibase+is+i].y += inv4PI*hostAccel[im].y;
          bodyAccel[ibase+is+i].z += inv4PI*hostAccel[im].z;
        }
        iblok++;
      }
    }
  }

  free(hostConstant);
  free(hostOffset);
  free(hostPosTarget);
  free(hostLnmSource);
  free(hostAccel);

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("l2p hipMalloc : %f s\n",t[1]);
//  printf("l2p hipMemcpy : %f s\n",t[2]);
//  printf("l2p cudaKernel : %f s\n",t[3]);
//  printf("l2p other      : %f s\n",t[0]);
//  printf("l2p flops      : %f G\n",flops/1e9);
  tic=flops;
}

// m2p
void FmmKernel::m2p(int numBoxIndex, int numLevel) {
  int nicall,jc,jj,ii,njd,ij,icall,jcall,iblok,im,jjd,jb,j,ibase,isize,is,i,ijc,jjdd;
  int ni,nj,nflop,*jbase,*jsize,*njcall,*njj,**interactionListOffsetStart,**interactionListOffsetEnd;
  vec3<int> boxIndex3D;
  const int offsetStride = 4*maxM2LInteraction+1;
  double tic,toc,flops,t[10],boxSize,op=0;

  for(i=0;i<10;i++) t[i]=0;
  tic=get_gpu_time();

  boxSize = rootBoxSize/(1 << numLevel);

  hostConstantSize=sizeof(float)*4;
  hostOffsetSize=sizeof(int)*targetBufferSize/threadsPerBlockTypeB*offsetStride;
  hostPosTargetSize=sizeof(float3)*targetBufferSize;
  hostMnmSourceSize=sizeof(float)*2*sourceBufferSize;
  hostAccelSize=sizeof(float3)*targetBufferSize;

  hostConstant=(float *)malloc(hostConstantSize);
  hostOffset=(int *)malloc(hostOffsetSize);
  hostPosTarget=(float3 *)malloc(hostPosTargetSize);
  hostMnmSource=(float *)malloc(hostMnmSourceSize);
  hostAccel=(float3 *)malloc(hostAccelSize);

  interactionListOffsetStart = new int* [maxM2LInteraction];
  for( i=0; i<maxM2LInteraction; i++ ) interactionListOffsetStart[i] = new int [numBoxIndexLeaf];
  interactionListOffsetEnd = new int* [maxM2LInteraction];
  for( i=0; i<maxM2LInteraction; i++ ) interactionListOffsetEnd[i] = new int [numBoxIndexLeaf];
  jbase = new int [numBoxIndexLeaf];
  jsize = new int [numBoxIndexLeaf];
  njcall = new int [numBoxIndexLeaf];
  njj = new int [numBoxIndexLeaf];

  hostConstant[0]=(float) boxSize;
  hostConstant[1]=(float) boxMin.x;
  hostConstant[2]=(float) boxMin.y;
  hostConstant[3]=(float) boxMin.z;

  if (is_set==0) {
    CUDA_SAFE_CALL(hipSetDevice(0));
    is_set=1;
  }
  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  if (hostOffsetSize>deviceOffsetSize) {
    if(deviceOffsetSize!=0) CUDA_SAFE_CALL(hipFree(deviceOffset));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceOffset,hostOffsetSize));
    deviceOffsetSize=hostOffsetSize;
  }
  if (hostPosTargetSize>devicePosTargetSize) {
    if(devicePosTargetSize!=0) CUDA_SAFE_CALL(hipFree(devicePosTarget));
    CUDA_SAFE_CALL(hipMalloc((void**)&devicePosTarget,hostPosTargetSize));
    devicePosTargetSize=hostPosTargetSize;
  }
  if (hostMnmSourceSize>deviceMnmSourceSize) {
    if(deviceMnmSourceSize!=0) CUDA_SAFE_CALL(hipFree(deviceMnmSource));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceMnmSource,hostMnmSourceSize));
    deviceMnmSourceSize=hostMnmSourceSize;
  }
  if (hostAccelSize>deviceAccelSize) {
    if(deviceAccelSize!=0) CUDA_SAFE_CALL(hipFree(deviceAccel));
    CUDA_SAFE_CALL(hipMalloc((void**)&deviceAccel,hostAccelSize));
    deviceAccelSize=hostAccelSize;
  }

  toc=tic;
  tic=get_gpu_time();
  t[1]+=tic-toc;

  ni = 0;
  nj = 0;
  nicall = 0;
  boxOffsetStart[0] = 0;
  jc = 0;
  for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
  for( ii=0; ii<numBoxIndex; ii++ ) {
    if( numInteraction[ii] != 0 ) {
      njd = 0;
      jc = 0;
      interactionListOffsetStart[0][ii] = 0;
      for( ij=0; ij<numInteraction[ii]; ij++ ) {
        jj = interactionList[ii][ij];
        if( njj[jj] == 0 ) {
          nj += numCoefficients;
          njj[jj] = 1;
        }
        njd += numCoefficients;
        if( njd > sourceBufferSize ) {
          interactionListOffsetEnd[jc][ii] = ij-1;
          jc++;
          interactionListOffsetStart[jc][ii] = ij;
          njd = numCoefficients;
        }
      }
      interactionListOffsetEnd[jc][ii] = numInteraction[ii]-1;
      ni += ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeB)/threadsPerBlockTypeB+1)
            *threadsPerBlockTypeB;
      if( jc != 0 ) {
        if( ii > boxOffsetStart[nicall] ) {
          njcall[nicall] = 1;
          boxOffsetEnd[nicall] = ii-1;
          nicall++;
          assert( nicall < numBoxIndexLeaf );
          boxOffsetStart[nicall] = ii;
          for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
        }
        if( ii != numBoxIndex ) {
          njcall[nicall] = jc+1;
          boxOffsetEnd[nicall] = ii;
          nicall++;
          assert( nicall < numBoxIndexLeaf );
          boxOffsetStart[nicall] = ii+1;
          for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
          ni = 0;
          nj = 0;
        }
      } else if ( ni > targetBufferSize || nj > sourceBufferSize ) {
        njcall[nicall] = jc+1;
        boxOffsetEnd[nicall] = ii-1;
        nicall++;
        assert( nicall < numBoxIndexLeaf );
        boxOffsetStart[nicall] = ii;
        for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
        ni = ((particleOffset[1][ii]-particleOffset[0][ii]+threadsPerBlockTypeB)/threadsPerBlockTypeB+1)
             *threadsPerBlockTypeB;
        nj = 0;
        for( ij=0; ij<numInteraction[ii]; ij++ ) {
          jj = interactionList[ii][ij];
          nj += numCoefficients;
          njj[jj] = 1;
        }
      }
    }
  }
  njcall[nicall] = jc+1;
  boxOffsetEnd[nicall] = numBoxIndex-1;
  if(numBoxIndex != 0) nicall++;

  for( icall=0; icall<nicall; icall++ ) {
    for( jcall=0; jcall<njcall[icall]; jcall++ ) {
      iblok = 0;
      jc = 0;
      jjd = 0;
      for( jj=0; jj<numBoxIndexLeaf; jj++ ) njj[jj] = 0;
      for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
        if( numInteraction[ii] != 0 ) {
          for( ij=interactionListOffsetStart[jcall][ii]; ij<=interactionListOffsetEnd[jcall][ii]; ij++ ) {
            jj = interactionList[ii][ij];
            jb = jj+levelOffset[numLevel-1];
            if( njj[jj] == 0 ) {
              jbase[jjd] = jc;
              for( j=0; j<numCoefficients; j++ ) {
                hostMnmSource[2*jc+0] = std::real(Mnm[jb][j]);
                hostMnmSource[2*jc+1] = std::imag(Mnm[jb][j]);
                jc++;
              }
              jsize[jjd] = jc-jbase[jjd];
              jjd++;
              njj[jj] = jjd;
            }
          }
          ibase = particleOffset[0][ii];
          isize = particleOffset[1][ii]-ibase+1;
          for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
            for( i=0; i<std::min(isize-is,threadsPerBlockTypeB); i++ ) {
              im = iblok*threadsPerBlockTypeB+i;
              hostPosTarget[im] = *(float3*) &bodyPos[ibase+is+i];
            }
            for( i=isize-is; i<threadsPerBlockTypeB; i++ ) {
              im = iblok*threadsPerBlockTypeB+i;
              hostPosTarget[im].x = 0;
              hostPosTarget[im].y = 0;
              hostPosTarget[im].z = 0;
            }
            hostOffset[iblok*offsetStride] = interactionListOffsetEnd[jcall][ii]
                                            -interactionListOffsetStart[jcall][ii]+1;
            ijc = 0;
            for( ij=interactionListOffsetStart[jcall][ii]; ij<=interactionListOffsetEnd[jcall][ii]; ij++ ) {
              jj = interactionList[ii][ij];
              jb = jj+levelOffset[numLevel-1];
              if( njj[jj] != 0 ) {
                jjdd = njj[jj]-1;
                tree.unmorton(boxIndexFull[jb],boxIndex3D);
                hostOffset[iblok*offsetStride+4*ijc+1] = jbase[jjdd];
                hostOffset[iblok*offsetStride+4*ijc+2] = boxIndex3D.x;
                hostOffset[iblok*offsetStride+4*ijc+3] = boxIndex3D.y;
                hostOffset[iblok*offsetStride+4*ijc+4] = boxIndex3D.z;
                op += (double) threadsPerBlockTypeB*jsize[jjdd];
                ijc++;
              }
            }
            iblok++;
          }
        }
      }

      if( iblok != 0 ) {
        toc=tic;
        tic=get_gpu_time();
        t[0]+=tic-toc;
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(deviceConstant),hostConstant,hostConstantSize));
        CUDA_SAFE_CALL(hipMemcpy(deviceOffset,hostOffset,hostOffsetSize,hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(devicePosTarget,hostPosTarget,hostPosTargetSize,hipMemcpyHostToDevice));
        CUDA_SAFE_CALL(hipMemcpy(deviceMnmSource,hostMnmSource,hostMnmSourceSize,hipMemcpyHostToDevice));
        toc=tic;
        tic=get_gpu_time();
        t[2]+=tic-toc;

        dim3 block(threadsPerBlockTypeB);
        dim3 grid(iblok);
        m2p_kernel<<< grid, block >>>(deviceOffset,devicePosTarget,deviceMnmSource,deviceAccel);
        cudaCheckError();
        nflop = 56;

        toc=tic;
        tic=get_gpu_time();
        t[3]+=tic-toc;
        CUDA_SAFE_CALL(hipMemcpy(hostAccel,deviceAccel,hostAccelSize,hipMemcpyDeviceToHost));
        toc=tic;
        tic=get_gpu_time();
        t[2]+=tic-toc;

      }

      iblok = 0;
      for( ii=boxOffsetStart[icall]; ii<=boxOffsetEnd[icall]; ii++ ) {
        if( numInteraction[ii] != 0 ) {
          ibase = particleOffset[0][ii];
          isize = particleOffset[1][ii]-ibase+1;
          for( is=0; is<isize; is+=threadsPerBlockTypeB ) {
            for( i=0; i<std::min(isize-is,threadsPerBlockTypeB); i++ ) {
              im = iblok*threadsPerBlockTypeB+i;
              bodyAccel[ibase+is+i].x += inv4PI*hostAccel[im].x;
              bodyAccel[ibase+is+i].y += inv4PI*hostAccel[im].y;
              bodyAccel[ibase+is+i].z += inv4PI*hostAccel[im].z;
            }
            iblok++;
          }
        }
      }
    }
  }
  free(hostOffset);
  free(hostPosTarget);
  free(hostMnmSource);
  free(hostAccel);
  for( i=0; i<maxM2LInteraction; i++ ) delete[] interactionListOffsetStart[i];
  delete[] interactionListOffsetStart;
  for( i=0; i<maxM2LInteraction; i++ ) delete[] interactionListOffsetEnd[i];
  delete[] interactionListOffsetEnd;
  delete[] jbase;
  delete[] jsize;
  delete[] njcall;
  delete[] njj;

  toc=tic;
  tic=get_gpu_time();
  t[0]+=tic-toc;
  for(i=0;i<9;i++) t[9]+=t[i];
  flops=op*((double)nflop)/t[9];
//  printf("m2p hipMalloc : %f s\n",t[1]);
//  printf("m2p hipMemcpy : %f s\n",t[2]);
//  printf("m2p cudaKernel : %f s\n",t[3]);
//  printf("m2p other      : %f s\n",t[0]);
//  printf("m2p flops      : %f G\n",flops/1e9);
  tic=flops;
}

